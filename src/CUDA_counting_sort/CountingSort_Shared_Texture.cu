#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <unistd.h>
#include <stdlib.h>

#define DATA int

#define START \
  hipEvent_t start,stop;\
  hipEventCreate(&start);\
  hipEventCreate(&stop);\
  hipEventRecord(start,0);\

#define STOP \
  hipEventRecord(stop,0);\
  hipEventSynchronize(stop);\
  float elapsed;\
  hipEventElapsedTime(&elapsed,start,stop);\
  elapsed/=1000.f;\
  hipEventDestroy(start);\
  hipEventDestroy(stop);\
  printf("Kernel elapsed time %fs \n", elapsed);

#define START_GF \
  hipEventCreate(&start_gf);\
  hipEventCreate(&stop_gf);\
  hipEventRecord(start_gf,0);\

#define STOP_GF \
  hipEventRecord(stop_gf,0);\
  hipEventSynchronize(stop_gf);\
  hipEventElapsedTime(&gf_elapsed,start_gf,stop_gf);\
  gf_elapsed/=1000.f;\
  hipEventDestroy(start_gf);\
  hipEventDestroy(stop_gf);\
  double dNumOps = (2 * N);\
  printf("Num ope: %f\n", dNumOps);\
  printf("Gflop time: %f\n", gf_elapsed);\
  gflops = 1.0e-9 * dNumOps/gf_elapsed;\
  printf("Gflops: %f\n", gflops);\

void make_csv(float time, int N, int gridsize, int thread_per_block){
  FILE* fp;
  char root_filename[] = "CS_shared_texture";

  char* filename = (char*) malloc(sizeof(char) * (strlen(root_filename) + 10*sizeof(char)));
  sprintf(filename,"%s_CountingSort_%d_v_%d_b_%d_tpb.csv",root_filename,N,gridsize,thread_per_block);
  
  if ( access( filename, F_OK ) == 0 ) {
     fp = fopen(filename,"a"); 

  } else {
     fp = fopen(filename,"w");
     fprintf(fp, "N; BlockSize; GridSize; time_sec\n");
  }
  fprintf(fp, "%d; %d; %d; %f\n", N, thread_per_block, gridsize, time);
  fclose(fp);
}

texture<DATA,1> text_mem;

__global__ void occurrence(DATA *C,int N,DATA val){
  extern __shared__ DATA cache[];  
  //__shared__ DATA cache[threads_per_block];
  //Creo un index per l'iterazione
  int idx_basic = blockIdx.x * blockDim.x + threadIdx.x;
  //Index associato alla cache
  int cacheIndex = threadIdx.x;

  //Creo una variabile contatore
  int occ_counter = 0;

  //Creo un index per l'iterazione
  int idx_iter = idx_basic;

  //Calcolo tutte le occorrenze
  while (idx_iter < N) {
    if(idx_iter < N){
      if(tex1Dfetch(text_mem,idx_iter) == val){
        occ_counter += 1;
      }
    }
    idx_iter += blockDim.x * gridDim.x;
  }

  //Le associo al vettore in memoria cache
  cache[cacheIndex] = occ_counter;

  //Sincronizzo i thread
  __syncthreads();

  //Eseguo la somma in parallelo
  int i = blockDim.x/2;
  while(i != 0){
      if(cacheIndex < i){
          cache[cacheIndex] = cache[cacheIndex]+cache[cacheIndex+i];
      }
      __syncthreads();
      i = i/2;
  }

  //associo al vettore in memoria globale quello che ho 
  //in prima posizione in memoria cache (risultato della somma)
  if(cacheIndex ==0){
    C[blockIdx.x] = cache[0];
  }
}

int main(int argc,char ** argv){
  if(argc<4) {
    fprintf(stderr,"ERROR too few arguments of: %s\n",argv[0]);
    exit(1);
  }

  DATA *A,*C,*devA,*devC;
  DATA max,min;

  int N = atoi(argv[1]);
  
  //Assegno il numero dei blocchi
  int blocks = atoi(argv[2]);
  dim3 dimGrid(blocks);

  //Assegno il numero dei thread per blocco
  int th_p_block = atoi(argv[3]);
  dim3 dimBlock(th_p_block);

  //alloco il vettore da ordinare su host e device
  A = (DATA*) malloc(N*sizeof(DATA));
  hipMalloc((void**)&devA,N*sizeof(DATA));

  //inizializzo il vettore
  for(int i=0;i<N;i++){
      A[i] = (N-i)%100;
  }

  //calcolo il massimo e il minimo
  max = A[0];
  min = A[0];
  for(int i=1;i<N;i++){
      if(A[i] > max){
          max = A[i];
      }else if(A[i] < min){
          min = A[i];
      }
  }

  //alloco il vettore di supporto su host e device
  C = (DATA*) malloc(blocks*sizeof(DATA));
  hipMalloc((void**)&devC,blocks*sizeof(DATA));

  //trasferisco i dati sul device
  hipMemcpy(devA,A,N*sizeof(DATA),hipMemcpyHostToDevice);
  hipChannelFormatDesc channel = hipCreateChannelDesc<DATA>();
  hipBindTexture(0,text_mem,devA,channel);

  int occorrenze = 0;
  int index_sorted = 0;
  START
  for(int i_occurr = min; i_occurr <= max; i_occurr++){
    //Per ogni elemento che va da min a max 
      //vado a cercarlo tramite chiamata a kernel
      occurrence<<<dimGrid,dimBlock,th_p_block*sizeof(DATA)>>>(devC,N,i_occurr);
      hipDeviceSynchronize();
      hipMemcpy(C,devC,blocks*sizeof(DATA),hipMemcpyDeviceToHost);

      //Calcolo le occorrenze restituite dal kernel
      for(int i=0;i<blocks;i++){
        occorrenze += C[i];
        for(int j=0;j<C[i];j++){
          A[index_sorted] = i_occurr;
          index_sorted++;
        }
      }
  }
  STOP
  hipUnbindTexture(text_mem);

  make_csv(elapsed, N, blocks, th_p_block);
  
  int ordinato = 1;
  for(int i=0;i<N;i++){
      if(i < N-2){
        if(A[i] > A[i+1]){
          ordinato = 0;
        }
      }
  }
  printf("\noccorrenze trovate: %d, occorrenze calcolate: %d\n",index_sorted,occorrenze);
  printf("ordinato = %d\n",ordinato);

  //libero la memoria
  free(A);
  free(C);
  hipFree(devA);
  hipFree(devC);
  
  return 0;
}